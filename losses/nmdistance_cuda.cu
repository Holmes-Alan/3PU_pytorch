#include "hip/hip_runtime.h"
#include <vector>
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_utils.h"


// closest point from xyz (b, n, 3) to xyz2 (b, m, 3)
template <unsigned int batch>
__global__ void NmDistanceKernel(int b, int n,
		const float * xyz, int m, const float * xyz2, float *result, int *result_i){
	// buffer to cache xyz2 4bytes*3*2^9 = 6kB shared
	__shared__ float buf[batch*3];
	// in total, loop through all b point clouds
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		// loop through the points in current point cloud in xyz2
		for (int k2=0; k2<m; k2+=batch){
			// fill the buffer with chunks of xyz2 values
			int end_k=min(m, k2+batch)-k2;
			for (int j=threadIdx.x; j<end_k*3; j+=blockDim.x){
				buf[j]=xyz2[(i*m+k2)*3+j];
			}
			__syncthreads();
			// loop through all n points in xyz
			for (int j=threadIdx.x; j<n; j+=blockDim.x){
				// current point in xyz
				float x1=xyz[(i*n+j)*3+0];
				float y1=xyz[(i*n+j)*3+1];
				float z1=xyz[(i*n+j)*3+2];
				int best_i=0;
				float best=0;
				// end_k&3 (0~3). end_ka largest multiple of 4
				int end_ka=end_k-(end_k&3);
				// loop through all buffered (xyz2)
				// find shortest distance from a point in xyz -> buffered xyz2
				// process four points in a thread
				if (end_ka==batch){
					for (int k=0;k<batch;k+=4){
						{
							float x2=buf[k*3+0]-x1;
							float y2=buf[k*3+1]-y1;
							float z2=buf[k*3+2]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
							float x2=buf[k*3+3]-x1;
							float y2=buf[k*3+4]-y1;
							float z2=buf[k*3+5]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
							float x2=buf[k*3+6]-x1;
							float y2=buf[k*3+7]-y1;
							float z2=buf[k*3+8]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
							float x2=buf[k*3+9]-x1;
							float y2=buf[k*3+10]-y1;
							float z2=buf[k*3+11]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}else{
					for (int k=0;k<end_ka;k+=4){
						{
							float x2=buf[k*3+0]-x1;
							float y2=buf[k*3+1]-y1;
							float z2=buf[k*3+2]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
							float x2=buf[k*3+3]-x1;
							float y2=buf[k*3+4]-y1;
							float z2=buf[k*3+5]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
							float x2=buf[k*3+6]-x1;
							float y2=buf[k*3+7]-y1;
							float z2=buf[k*3+8]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
							float x2=buf[k*3+9]-x1;
							float y2=buf[k*3+10]-y1;
							float z2=buf[k*3+11]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}
				for (int k=end_ka;k<end_k;k++){
					float x2=buf[k*3+0]-x1;
					float y2=buf[k*3+1]-y1;
					float z2=buf[k*3+2]-z1;
					float d=x2*x2+y2*y2+z2*z2;
					if (k==0 || d<best){
						best=d;
						best_i=k+k2;
					}
				}
				if (k2==0 || result[(i*n+j)]>best){
					result[(i*n+j)]=best;
					result_i[(i*n+j)]=best_i;
				}
			}
			__syncthreads();
		}
	}
}

std::vector<at::Tensor> NmDistanceKernelLauncher(int b,int n, at::Tensor xyz,
		int m, at::Tensor xyz2, at::Tensor result, at::Tensor result_i, at::Tensor result2, at::Tensor result2_i){
	// bxn
	unsigned int n_threads, n_blocks;
	n_threads = opt_n_threads(n);
	n_blocks = min(32, (n*b + n_threads/2)/n_threads);
	switch (n_threads) {
		case 512:
		NmDistanceKernel<512><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 256:
		NmDistanceKernel<256><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 128:
		NmDistanceKernel<128><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 64:
		NmDistanceKernel<64><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 32:
		NmDistanceKernel<32><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 16:
		NmDistanceKernel<16><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 8:
		NmDistanceKernel<8><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 4:
		NmDistanceKernel<4><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 2:
		NmDistanceKernel<2><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		case 1:
		NmDistanceKernel<1><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
		default:
		NmDistanceKernel<512><<<n_blocks, n_threads>>>(b,n,xyz.data<float>(),
			m,xyz2.data<float>(),result.data<float>(),result_i.data<int32_t>());
		break;
	}

	// bxm
	n_threads = opt_n_threads(m);
	n_blocks = min(32, (m*b + n_threads/2)/n_threads);
	switch (n_threads){
		case 512:
		NmDistanceKernel<512><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 256:
		NmDistanceKernel<256><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 128:
		NmDistanceKernel<128><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 64:
		NmDistanceKernel<64><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 32:
		NmDistanceKernel<32><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 16:
		NmDistanceKernel<16><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 8:
		NmDistanceKernel<8><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 4:
		NmDistanceKernel<4><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 2:
		NmDistanceKernel<2><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		case 1:
		NmDistanceKernel<1><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
		default:
		NmDistanceKernel<512><<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz.data<float>(),result2.data<float>(),result2_i.data<int32_t>());
		break;
	}

	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd updateOutput: %s\n", hipGetErrorString(err));
	    exit(-1);
	  }
	return {result, result_i, result2, result2_i};
}

// template std::vector<at::Tensor> NmDistanceKernelLauncher<double>(int b,int n, at::Tensor xyz,
// 		int m, at::Tensor xyz2, at::Tensor result, at::Tensor result_i, at::Tensor result2, at::Tensor result2_i);
// template std::vector<at::Tensor> NmDistanceKernelLauncher<float>(int b,int n, at::Tensor xyz,
// 		int m, at::Tensor xyz2, at::Tensor result, at::Tensor result_i, at::Tensor result2, at::Tensor result2_i);

__global__ void NmDistanceGradKernel(int b, int n, const float * xyz1,
		int m, const float * xyz2, const float * grad_dist1, const int * idx1, float * grad_xyz1, float * grad_xyz2){
	for (int i=blockIdx.x; i<b; i+=gridDim.x){
		for (int j=threadIdx.x; j<n; j+=blockDim.x){
			// j-th point in xyz1 is mapped to j2-th point in xyz2
			float x1=xyz1[(i*n+j)*3+0];
			float y1=xyz1[(i*n+j)*3+1];
			float z1=xyz1[(i*n+j)*3+2];
			int j2=idx1[i*n+j];
			float x2=xyz2[(i*m+j2)*3+0];
			float y2=xyz2[(i*m+j2)*3+1];
			float z2=xyz2[(i*m+j2)*3+2];
			float g=grad_dist1[i*n+j]*2;
			atomicAdd(grad_xyz1+(i*n+j)*3+0, g*(x1-x2));
			atomicAdd(grad_xyz1+(i*n+j)*3+1, g*(y1-y2));
			atomicAdd(grad_xyz1+(i*n+j)*3+2, g*(z1-z2));
			atomicAdd(grad_xyz2+(i*m+j2)*3+0, -(g*(x1-x2)));
			atomicAdd(grad_xyz2+(i*m+j2)*3+1, -(g*(y1-y2)));
			atomicAdd(grad_xyz2+(i*m+j2)*3+2, -(g*(z1-z2)));
		}
	}
}

__global__ void NmDistanceGrad1Kernel(int b, int n, const float * xyz1,
		int m, const float * xyz2, const float * grad_dist1, const int * idx1, float * grad_xyz1){
	for (int i=blockIdx.x; i<b; i+=gridDim.x){
		for (int j=threadIdx.x; j<n; j+=blockDim.x){
			// j-th point in xyz1 is mapped to j2-th point in xyz2
			float x1=xyz1[(i*n+j)*3+0];
			float y1=xyz1[(i*n+j)*3+1];
			float z1=xyz1[(i*n+j)*3+2];
			int j2=idx1[i*n+j];
			float x2=xyz2[(i*m+j2)*3+0];
			float y2=xyz2[(i*m+j2)*3+1];
			float z2=xyz2[(i*m+j2)*3+2];
			float g=grad_dist1[i*n+j]*2;
			atomicAdd(grad_xyz1+(i*n+j)*3+0, g*(x1-x2));
			atomicAdd(grad_xyz1+(i*n+j)*3+1, g*(y1-y2));
			atomicAdd(grad_xyz1+(i*n+j)*3+2, g*(z1-z2));
		}
	}
}

__global__ void NmDistanceGrad2Kernel(int b, int n, const float * xyz1,
		int m, const float * xyz2, const float * grad_dist1, const int * idx1, float * grad_xyz2){
	for (int i=blockIdx.x; i<b; i+=gridDim.x){
		for (int j=threadIdx.x; j<n; j+=blockDim.x){
			// j-th point in xyz1 is mapped to j2-th point in xyz2
			float x1=xyz1[(i*n+j)*3+0];
			float y1=xyz1[(i*n+j)*3+1];
			float z1=xyz1[(i*n+j)*3+2];
			int j2=idx1[i*n+j];
			float x2=xyz2[(i*m+j2)*3+0];
			float y2=xyz2[(i*m+j2)*3+1];
			float z2=xyz2[(i*m+j2)*3+2];
			float g=grad_dist1[i*n+j]*2;
			atomicAdd(grad_xyz2+(i*m+j2)*3+0, -(g*(x1-x2)));
			atomicAdd(grad_xyz2+(i*m+j2)*3+1, -(g*(y1-y2)));
			atomicAdd(grad_xyz2+(i*m+j2)*3+2, -(g*(z1-z2)));
		}
	}
}

std::vector<at::Tensor> NmDistanceGradKernelLauncher(int b,int n, at::Tensor xyz1,
		int m, at::Tensor xyz2,
		at::Tensor grad_dist1, at::Tensor idx1,
		at::Tensor grad_dist2, at::Tensor idx2,
		bool requires_grad_1, bool requires_grad_2,
		at::Tensor grad_xyz1, at::Tensor grad_xyz2) {
	std::vector<at::Tensor> v;
	unsigned int n_threads, n_blocks;
	if (!requires_grad_2)
	{
		n_threads = opt_n_threads(n);
		n_blocks = min(32, (n*b + n_threads/2)/n_threads);
		NmDistanceGrad1Kernel<<<n_blocks, n_threads>>>(b,n,xyz1.data<float>(),
			m,xyz2.data<float>(),grad_dist1.data<float>(),idx1.data<int32_t>(),grad_xyz1.data<float>());
		n_threads = opt_n_threads(m);
		n_blocks = min(32, (m*b + n_threads/2)/n_threads);
		NmDistanceGrad2Kernel<<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz1.data<float>(),grad_dist2.data<float>(),idx2.data<int32_t>(),grad_xyz1.data<float>());
		v = {grad_xyz1};
	}
	if (!requires_grad_1)
	{
		n_threads = opt_n_threads(m);
		n_blocks = min(32, (m*b + n_threads/2)/n_threads);
		NmDistanceGrad2Kernel<<<n_blocks, n_threads>>>(b,n,xyz1.data<float>(),
			m,xyz2.data<float>(),grad_dist1.data<float>(),idx2.data<int32_t>(),grad_xyz1.data<float>());
		n_threads = opt_n_threads(n);
		n_blocks = min(32, (n*b + n_threads/2)/n_threads);
		NmDistanceGrad1Kernel<<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz1.data<float>(),grad_dist2.data<float>(),idx1.data<int32_t>(),grad_xyz2.data<float>());
		v = {grad_xyz1};
	}
	if (requires_grad_1 && requires_grad_2)
	{
		n_threads = opt_n_threads(n);
		n_blocks = min(32, (n*b + n_threads/2)/n_threads);
		NmDistanceGradKernel<<<n_blocks, n_threads>>>(b,n,xyz1.data<float>(),
			m,xyz2.data<float>(),grad_dist1.data<float>(),idx1.data<int32_t>(),grad_xyz1.data<float>(), grad_xyz2.data<float>());

		n_threads = opt_n_threads(m);
		n_blocks = min(32, (m*b + n_threads/2)/n_threads);
		NmDistanceGradKernel<<<n_blocks, n_threads>>>(b,m,xyz2.data<float>(),
			n,xyz1.data<float>(),grad_dist2.data<float>(),idx2.data<int32_t>(),grad_xyz2.data<float>(),grad_xyz1.data<float>());
		v = {grad_xyz1, grad_xyz2};
	}

	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd get grad: %s\n", hipGetErrorString(err));
	    exit(-1);
	  }
	return v;
}

